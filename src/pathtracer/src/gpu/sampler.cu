#include "hip/hip_runtime.h"
#include "sampler.h"

namespace VRRT {

// Uniform Sampler2D Implementation //


__device__
Vector2D uniformGridSample(hiprandState *state) {
  return Vector2D(hiprand_uniform(state), hiprand_uniform(state));
}

// Uniform Hemisphere Sampler3D Implementation //
__device__
Vector3D uniformHemisphereSample(hiprandState *state) {

  float Xi1 = hiprand_uniform(state);
  float Xi2 = hiprand_uniform(state);

  float theta = acos(Xi1);
  float phi = 2.0 * PI * Xi2;

  float xs = sinf(theta) * cosf(phi);
  float ys = sinf(theta) * sinf(phi);
  float zs = cosf(theta);

  return Vector3D::make(xs, ys, zs);

}

__device__
Vector3D cosineWeightedHemisphereSample(hiprandState *state)
{
  float f;
  return cosineWeightedHemisphereSample(state, &f);
}

__device__
Vector3D cosineWeightedHemisphereSample(hiprandState *state, float *pdf) {
  float z1 = hiprand_uniform(state), z2 = hiprand_uniform(state);
  float theta = 2 * PI * z1, r = sqrtf(z2), z = sqrtf(1 - r*r);
  *pdf = z / PI;
  return Vector3D::make(r * cosf(theta), r * sinf(theta), z);
}


} // namespace CMU462
