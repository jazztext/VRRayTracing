#include "hip/hip_runtime.h"
#include "ray.h"
#include "bvhGPU.h"
#include "../bvh.h"
#include "../cycleTimer.h"
#include "pathtracer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

namespace VRRT {

extern __constant__ constantParams cuGlobals;

__global__ void initCurand2(hiprandState *state)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int ind = y * gridDim.x * blockDim.x + x;
  hiprand_init(1234, ind, 0, &state[ind]);
}

__device__ Vector3D pointInBox(BBox bbox, hiprandState *state)
{
  float x = hiprand_uniform(state),
        y = hiprand_uniform(state),
        z = hiprand_uniform(state);
  Vector3D p;
  p = Vector3D::make(x * bbox.extent.v.x, y * bbox.extent.v.y, z * bbox.extent.v.z);
  return p + bbox.min;
}

__global__ void genRays(Ray *rays, int numRays, hiprandState *states, BBox bbox)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numRays) return;
  hiprandState localState = states[i];
  Vector3D p1 = pointInBox(bbox, &localState);
  Vector3D p2 = pointInBox(bbox, &localState);
  Vector3D d = p2 - p1;
  Vector3D o = p1 - bbox.extent.norm() * d;
  rays[i] = Ray(o, d);
}

__global__ void raycast(Ray *rays, int numRays, BVHGPU bvh)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numRays) return;
  Ray r = rays[i];
  Intersection inters;
  bvh.intersect(r, &inters);
}

void benchmark(CMU462::StaticScene::BVHAccel *bvh, int numRays)
{
  Vector3D *points, *normals;
  BVHGPU bvhGPU(bvh, &points, &normals);
  constantParams params;
  params.points = points;
  params.normals = normals;
  cudaCheckError( hipMemcpyToSymbol(HIP_SYMBOL(cuGlobals), &params, sizeof(constantParams)) );

  dim3 blockDim(256);
  dim3 gridDim((numRays + blockDim.x - 1) / blockDim.x);

  //init hiprand state
  hiprandState *states;
  int stateSize = blockDim.x * blockDim.y * gridDim.x * gridDim .y;
  cudaCheckError( hipMalloc(&states, sizeof(hiprandState) * stateSize) );
  initCurand2<<<gridDim, blockDim>>>(states);
  cudaCheckError( hipGetLastError() );
  cudaCheckError( hipDeviceSynchronize() );

  //generate rays
  Ray *rays;
  cudaCheckError( hipMalloc(&rays, sizeof(Ray) * numRays) );
  CMU462::BBox bbox = bvh->get_bbox();
  BBox bbox2(bbox.min, bbox.max);
  genRays<<<gridDim, blockDim>>>(rays, numRays, states, bbox2);
  cudaCheckError( hipGetLastError() );
  cudaCheckError( hipDeviceSynchronize() );

  //run test
  std::cout << "Casting rays... ";
  fflush(stdout);
  hipProfilerStart();
  double start = CycleTimer::currentSeconds();
  raycast<<<gridDim, blockDim>>>(rays, numRays, bvhGPU);
  cudaCheckError( hipGetLastError() );
  cudaCheckError( hipDeviceSynchronize() );
  double end = CycleTimer::currentSeconds();
  hipProfilerStop();
  std::cout << "Done! (" << end - start << " sec)\n";

}

}
