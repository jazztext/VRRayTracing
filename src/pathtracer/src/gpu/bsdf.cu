#include "hip/hip_runtime.h"
#include "bsdf.h"
#include <iostream>

namespace VRRT {

__device__
void make_coord_space(Matrix3x3& o2w, const Vector3D& n) {

    Vector3D z = Vector3D::make(n.v.x, n.v.y, n.v.z);
    Vector3D h = z;
    if (fabsf(h.v.x) <= fabsf(h.v.y) && fabsf(h.v.x) <= fabsf(h.v.z)) h.v.x = 1.0;
    else if (fabsf(h.v.y) <= fabsf(h.v.x) && fabsf(h.v.y) <= fabsf(h.v.z)) h.v.y = 1.0;
    else h.v.z = 1.0;

    z.normalize();
    Vector3D y = cross(h, z);
    y.normalize();
    Vector3D x = cross(z, y);
    x.normalize();

    o2w[0] = x;
    o2w[1] = y;
    o2w[2] = z;

}

__device__
Spectrum BSDF::f(const Vector3D& wo, const Vector3D& wi)
{
  switch (t) {
    case DIFFUSE:
      return color * (1.f / PI);
    default:
      return Spectrum();
  }
}

__device__
Spectrum BSDF::sample_f (const Vector3D& wo, Vector3D* wi, float* pdf,
                         bool& inMat, hiprandState *state)
{
  switch (t) {
    case MIRROR:
      inMat = false;
      reflect(wo, wi);
      *pdf = 1;
      return color * (1 / wo.v.z);
    case GLASS:
      return glassSample(wo, wi, pdf, inMat, state);
    default:
      inMat = false;
      *wi = cosineWeightedHemisphereSample(state, pdf);
      return f(wo, *wi);
  }
}

__device__
Spectrum BSDF::get_emission()
{
  switch (t) {
    case EMISSION:
      return color * (1.0 / PI);
    default:
      return Spectrum();
  }
}

__device__
bool BSDF::is_delta()
{
  switch (t) {
    case MIRROR:
    case GLASS:
      return true;
    default:
      return false;
  }

}

__host__
BSDF *BSDF::copyToDev()
{
  BSDF *location;
  cudaCheckError( hipMalloc(&location, sizeof(BSDF)) );
  cudaCheckError( hipMemcpy(location, this, sizeof(BSDF),
                             hipMemcpyHostToDevice) );
  return location;
}

__device__
Spectrum BSDF::glassSample(const Vector3D& wo, Vector3D* wi, float* pdf,
                           bool& inMat, hiprandState *state) {

  // Compute Fresnel coefficient and either reflect or refract based on it.
  float ni, nt;
  if (inMat) {
    ni = ior;
    nt = 1;
  }
  else {
    ni = 1;
    nt = ior;
  }
  Vector3D transmit;
  bool tir = !refract(wo, &transmit, ior, inMat);
  float cosThetaT = -wo.v.z, cosThetaI = transmit.v.z;
  float rPar = (nt * cosThetaI - ni * cosThetaT) / (nt * cosThetaI + ni * cosThetaT);
  float rPerp = (ni * cosThetaI - nt * cosThetaT) / (ni * cosThetaI + nt * cosThetaT);
  float fr = .5 * (powf(rPar, 2) + powf(rPerp, 2));
  if (tir || hiprand_uniform(state) < fr) { //reflect
    reflect(wo, wi);
    *pdf = 1;
    return color2 * (1 / fabs(wo.v.z));
  }
  else { //refract
    *wi = transmit;
    *pdf = 1;
    inMat = !inMat;
    return color * powf(ni / nt, 2) * (1 / (fabsf(cosThetaI)));
  }
}

__device__
void BSDF::reflect(const Vector3D& wo, Vector3D* wi) {

  // Implement reflection of wo about normal (0,0,1) and store result in wi.
  *wi = wo;
  wi->v.x *= -1;
  wi->v.y *= -1;

}

__device__
bool BSDF::refract(const Vector3D& wo, Vector3D* wi, float ior, bool inMat) {

  // Use Snell's Law to refract wo surface and store result ray in wi.
  // Return false if refraction does not occur due to total internal reflection
  // and true otherwise. When dot(wo,n) is positive, then wo corresponds to a
  // ray entering the surface through vacuum.

  float ni, nt;
  if (!inMat) {
    ni = 1;
    nt = ior;
  }
  else {
    ni = ior;
    nt = 1;
  }
  float radicand = 1 - powf(ni / nt, 2) * (1 - powf(wo.v.z, 2));
  if (radicand < 0) return false; //total internal reflection
  wi->v.z = (wo.v.z > 0) ? -sqrtf(radicand) : sqrtf(radicand);
  float scale = sqrtf(1 - radicand) / sqrtf(powf(wo.v.x, 2) + powf(wo.v.y, 2));
  wi->v.x = -scale * wo.v.x;
  wi->v.y = -scale * wo.v.y;
  return true;

}

} // namespace CMU462
