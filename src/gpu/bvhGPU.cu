#include "bvhGPU.h"
#include <iostream>
#include <unordered_map>

namespace VRRT {

BVHNodeGPU *BVHGPU::flattenNode(CMU462::StaticScene::BVHNode *node,
                                std::vector<BVHNodeGPU> &flatNodes)
{
  flatNodes.emplace_back(node->bb, node->start, node->range);
  BVHNodeGPU *currNode = &flatNodes.back();
  if (!node->isLeaf()) {
    currNode->l = flattenNode(node->l, flatNodes);
    currNode->r = flattenNode(node->r, flatNodes);
  }
  return currNode;
}

BVHGPU::BVHGPU(CMU462::StaticScene::BVHAccel *bvh)
{
  std::vector<PrimitiveGPU> cpuPrims;
  std::unordered_map<BSDF *, BSDF *> bsdfs;

  //copy bsdfs and primitives to GPU
  for (int i = 0; i < bvh->primitives.size(); i++) {
    cpuPrims.push_back(bvh->primitives[i]->toGPU(bsdfs));
  }
  cudaCheckError( hipDeviceSynchronize() );
  cudaCheckError( hipMalloc(&primitives, sizeof(PrimitiveGPU) * cpuPrims.size()) );
  cudaCheckError( hipMemcpy(primitives, cpuPrims.data(), sizeof(PrimitiveGPU)*cpuPrims.size(),
             hipMemcpyHostToDevice) );

  //flatten node structure and copy to GPU
  std::vector<BVHNodeGPU> flatNodes;
  flattenNode(bvh->root, flatNodes);
  cudaCheckError( hipMalloc(&nodes, sizeof(BVHNodeGPU) * flatNodes.size()) );
  cudaCheckError( hipMemcpy(nodes, flatNodes.data(), sizeof(BVHNodeGPU) * flatNodes.size(),
             hipMemcpyHostToDevice) );
}

BVHGPU::~BVHGPU() {
  //cudaCheckError( hipFree(nodes) );
  //cudaCheckError( hipFree(primitives) );
}

__device__
BBox BVHGPU::get_bbox() const {
  return nodes[0].bb;
}

__device__
bool BVHGPU::intersect(Ray &ray) const {

  // TODO:
  // Implement ray - bvh aggregate intersection test. A ray intersects
  // with a BVH aggregate if and only if it intersects a primitive in
  // the BVH that is not an aggregate.

  Intersection i;

  return intersect(ray, &i);

}

__device__
bool BVHGPU::intersectNode(BVHNodeGPU *node, Ray& ray, Intersection *i) const
{
  if (node->isLeaf()) {
    bool hit = false;
    for (int n = node->start; n < node->start + node->range; n++) {
      if (primitives[n].intersect(ray, i)) hit = true;
    }
    return hit;
  }
  float minTL = ray.min_t, minTR = ray.min_t;
  float maxTL = ray.max_t, maxTR = ray.max_t;
  bool hitLeft = node->l->bb.intersect(ray, minTL, maxTL);
  bool hitRight = node->r->bb.intersect(ray, minTR, maxTR);
  BVHNodeGPU *first, *second;
  bool hitFirst, hitSecond;
  if (minTL < minTR) {
    first = node->l; second = node->r; hitFirst = hitLeft; hitSecond = hitRight;
  }
  else {
    first = node->r; second = node->l; hitFirst = hitRight; hitSecond = hitLeft;
  }
  bool hit = false;
  if (hitFirst && intersectNode(first, ray, i)) hit = true;
  if (hitSecond && minTR < ray.max_t && intersectNode(second, ray, i))
    hit = true;
  return hit;
}

__device__
bool BVHGPU::intersect(Ray &ray, Intersection *i) const {

  // TODO:
  // Implement ray - bvh aggregate intersection test. A ray intersects
  // with a BVH aggregate if and only if it intersects a primitive in
  // the BVH that is not an aggregate. When an intersection does happen.
  // You should store the non-aggregate primitive in the intersection data
  // and not the BVH aggregate itself.

  bool returnVal = intersectNode(&nodes[0], ray, i);
  return returnVal;

}

}
